#include "hip/hip_runtime.h"
﻿#include "Particle.h"
#include "Vector.h"
#include <fstream>
#include <iostream>
#include <string>
#include <filesystem>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

/*Solution of N-body problem with Particle-Particle (direct sum) method.
This program calculates in 3D space,
with nondimensialization (G = 1),
equations of motion are solved with Euler's method.*/


Particle* InitializeNBodySystem(const std::string path, int& n);
void SetInitialParameters(
	const std::string path,
	double& timeStep,
	double& cuttingRadius,
	double& limitOfLoop,
	int& consoleLogFrequency);

__global__ void calculateForce(Particle* particles, const size_t n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n)
	{
		Vector force;
		force.x = 0.0;
		force.y = 0.0;
		force.z = 0.0;

		for (int j = 0; j < n; ++j)
		{
			if (i != j)
			{
				double distanceX = particles[j].position.x - particles[i].position.x;
				double distanceY = particles[j].position.y - particles[i].position.y;
				double distanceZ = particles[j].position.z - particles[i].position.z;

				double vector = sqrt(distanceX * distanceX + distanceY * distanceY + distanceZ * distanceZ);
				double denominator = vector * vector * vector;

				force.x = force.x + distanceX * particles[i].mass * particles[j].mass / denominator;
				force.y = force.y + distanceY * particles[i].mass * particles[j].mass / denominator;
				force.z = force.z + distanceZ * particles[i].mass * particles[j].mass / denominator;
			}
		}

		particles[i].acceleration.x = force.x / particles[i].mass;
		particles[i].acceleration.y = force.y / particles[i].mass;
		particles[i].acceleration.z = force.z / particles[i].mass;
	}
}


int main()
{
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	int n;
	double timeStep;
	double cuttingRadius;
	double limitOfLoop;
	int consoleLogFrequency;

	SetInitialParameters("initialParameters.txt", timeStep, cuttingRadius, limitOfLoop, consoleLogFrequency);

	std::cout << "Reading input file\n";
	Particle* particles = InitializeNBodySystem("Particles.txt", n);
	std::cout << "File is read\n\n\n";

	Particle* particlesDevice;
	const size_t sizeBytes = n * sizeof(Particle);

	dim3 dimBlock(1024);
	dim3 dimGrid(n / 1024 + 1);


	std::filesystem::path path = L"coordinates";
	if (std::filesystem::exists(path))
	{
		std::filesystem::remove_all(path);
	}

	if (!std::filesystem::create_directory(path))
	{
		std::cout << "Error making a directory\n";
		return 1;
	}

	double time = 0.0;
	long count = 0;
	bool isConsoleLogSet = consoleLogFrequency > 0;
	for (;;)
	{
		if (isConsoleLogSet && count % consoleLogFrequency == 0)
			std::cout << count << " iterations have passed. Moment of time: " << time << "\n";

		std::ofstream fileCoordinates;
		std::string countStr = std::to_string(count);
		fileCoordinates.open("coordinates\\" + countStr + ".csv");
		fileCoordinates << "x;y;z\n";

		cudaStatus = hipMalloc((void**)&particlesDevice, sizeBytes);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			return 1;
		}

		cudaStatus = hipMemcpy(particlesDevice, particles, sizeBytes, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}

		std::cout << "\nWriting to file all positions\n";
		auto startWriting = std::chrono::high_resolution_clock::now();
		for (int i = 0; i < n; ++i)
		{
			fileCoordinates << particles[i].position << std::endl;
		}
		auto finishWriting = std::chrono::high_resolution_clock::now();
		auto msWriting = std::chrono::duration_cast<std::chrono::milliseconds>(finishWriting - startWriting);
		std::cout << "Writing to file has finished in " << msWriting.count() << " milliseconds\n\n";


		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		std::cout << "\nCalculating forces on GPU\n";
		hipEventRecord(start, 0);

		calculateForce<<<dimGrid, dimBlock>>> (particlesDevice, n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return -1;
		}

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		std::cout << "Calculating forces on GPU finished in " << elapsedTime << " milliseconds\n\n";

		hipEventDestroy(start);
		hipEventDestroy(stop);



		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return 1;
		}

		cudaStatus = hipMemcpy(particles, particlesDevice, sizeBytes, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}

		std::cout << "\nCalculating velocities and positions with Euler's method\n";
		auto startEuler = std::chrono::high_resolution_clock::now();
		for (int i = 0; i < n; ++i)
		{
			particles[i].velocity = particles[i].velocity + particles[i].acceleration * timeStep;

			particles[i].position = particles[i].position + particles[i].velocity * timeStep;
		}
		auto finishEuler = std::chrono::high_resolution_clock::now();
		auto msEuler = std::chrono::duration_cast<std::chrono::milliseconds>(finishEuler - startEuler);
		std::cout << "Calculating with Euler's method has finished in " << msEuler.count() << " milliseconds\n\n\n\n";

		time += timeStep;
		++count;
		fileCoordinates.close();
	}

	delete[] particles;
	std::system("pause");
	return 0;
}

Particle* InitializeNBodySystem(const std::string path, int& n)
{
	std::ifstream fileParticles;
	fileParticles.open(path);

	char tempString[256];
	fileParticles.getline(tempString, 256, ':');

	fileParticles >> n;
	Particle* particles = new Particle[n];

	fileParticles.get();
	fileParticles.get();

	fileParticles.getline(tempString, 256);

	for (int i = 0; i < n; ++i)
	{
		fileParticles >> particles[i].mass;
		fileParticles.get();
		fileParticles >> particles[i].velocity.x >> particles[i].velocity.y >> particles[i].velocity.z;
		fileParticles.get();
		fileParticles >> particles[i].position.x >> particles[i].position.y >> particles[i].position.z;
	}

	fileParticles.close();
	return particles;
}


void SetInitialParameters(
	const std::string path,
	double& timeStep,
	double& cuttingRadius,
	double& limitOfLoop,
	int& consoleLogFrequency)
	{
		std::ifstream initialFile;
		initialFile.open(path);

		char tempString[32];
		initialFile.getline(tempString, 32, ';');

		initialFile.getline(tempString, 32, '=');
		initialFile >> timeStep;

		initialFile.getline(tempString, 32, '=');
		initialFile >> cuttingRadius;

		initialFile.getline(tempString, 32, '=');
		initialFile >> limitOfLoop;

		initialFile.getline(tempString, 32, '=');
		initialFile >> consoleLogFrequency;

		initialFile.close();
	}
